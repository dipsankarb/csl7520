
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000

__device__ unsigned size;
__device__ int list[N];

__global__ void kernel() {
	int id=blockIdx.x*blockDim.x + threadIdx.x;

	list[atomicInc(&size,N)] = id;

}

__global__ void print_kernel() {
	int i;
	printf("CURRENT SIZE IS %d\n",size);
	for(i=0;i<size;i++)
		printf("list[%d] = %d\n",i,list[i]);
}

int main(){
	hipMemset(&size, 0, sizeof(int));

	kernel<<<4,64>>>();

	hipDeviceSynchronize();

	print_kernel<<<1,1>>>();

	hipDeviceSynchronize();
	return 0;
}
