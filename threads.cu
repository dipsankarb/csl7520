
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void deviceKernel(){

//	if(threadIdx.x ==0 && blockIdx.x==0 &&
//			threadIdx.y == 0 && blockIdx.y==0 &&
//			threadIdx.z==0 && blockIdx.z==0) {
	if(threadIdx.x==0){
		printf("%d %d %d %d %d %d\n",gridDim.x,gridDim.y,gridDim.z,blockDim.x,blockDim.y,blockDim.z);

	}
}

int main() {

	dim3 grid(2,3,4);
	dim3 block(5,6,7);

	deviceKernel<<<grid,block>>>();

	hipDeviceSynchronize();

	return 0;
}
