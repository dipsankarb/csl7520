

#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 1000000
#define BLOCK 1024

__global__ void init_kernel(int *a){
	int id;
	id=blockIdx.x*blockDim.x+threadIdx.x;

	if(id<N) 
		a[id]=id;
}

int main(){
	int *a,*d_a,i;
	int nblocks;

	hipMalloc(&d_a,N*sizeof(int));

	a=(int*)malloc(N*sizeof(int));

	nblocks=ceil(N/BLOCK);
	printf("NUMBER OF BLOCKS %d\n",nblocks);

	init_kernel<<<nblocks+1, BLOCK>>>(d_a);

	hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

	for(i=1; i<11 ; i++)
		printf("%d ", a[N-i]);

	return 0;
}
