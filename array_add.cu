#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 1000


__global__ void squaring(int *a) {
	a[threadIdx.x]=(threadIdx.x+1)*(threadIdx.x+1);
	printf("%s",msg);
}

int main() {
	int a[N],i;
	int *d_a;

	hipMalloc(&d_a, N*sizeof(int));

	for(i=0; i<N; i++)
		a[i]=i+1;

	hipMemcpy(d_a, a,  N*sizeof(int), hipMemcpyHostToDevice);

	squaring<<<1,N>>>(d_a);

	hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

	//COMPUTATIONS AFTER THE MEMCPY

	for(i=0; i<N; i++){
		printf("%d\n",a[i]);
	}

	hipFree(d_a);

	return 0;
}
