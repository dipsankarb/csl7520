
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 5
#define M 6

void matrix_init_cpu(int *a){

	int i,j,c=0;

	for(i=0; i<N; i++)
		for(j=0;j<M;j++)
			a[i*M+j] = c++;
}

__global__ void device_kernel(){
//	if(threadIdx.x == 0 && blockIdx.x==0 &&
//		threadIdx.y == 0 && blockIdx.y ==0 &&
//		threadIdx.z == 0 && blockIdx.z == 0)
	if(threadIdx.x == 0)
		printf("%d %d %d %d %d %d\n",gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
}

__global__ void matrix_init_gpu(int *a){
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	a[id]=id;
}

int main(){
	int i,j;
	//dim3 grid(2,3,4);

	//dim3 block(5,6,7);

	//device_kernel<<<grid,block>>>();

	dim3 block(N,M,1);

	int *a = (int*)malloc(N*M*sizeof(int));
	int *d_a;

	hipMalloc(&d_a, N*M*sizeof(int));

	//matrix_init_cpu(a);

	matrix_init_gpu<<<N, M>>>(d_a);

	hipMemcpy(a, d_a, N*M*sizeof(int), hipMemcpyDeviceToHost);

	for(i=0;i<N;i++){
		for(j=0;j<M;j++){
			printf("%d ",a[i*M+j]);
		}
		printf("\n");
	}

	hipDeviceSynchronize();

	return 0;
}
