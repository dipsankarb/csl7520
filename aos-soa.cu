
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct node1{
	int a;
	double b;
	char c;
};
struct node2{
	int a[N];
	double b[N];
	char c[N];
};

__global__ void dKernel1(node1 *arr1){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid>=N)
		return;

//	printf("WORKING ON AOS\n");
	arr1[tid].a=1+(tid*tid)/434324;
	arr1[tid].b=1.5;
	arr1[tid].c='c';

}

__global__ void dKernel2(node2 *arr2){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid>=N)
		return;

//	printf("WORKING ON SOA\n");
	arr2->a[tid]=1+(tid*tid)/32434423;
	arr2->b[tid]=1.5;
	arr2->c[tid]='c';
//	printf("KERNEL DONE\n");
}



int main(){
	int i,j,k;
	float ms;


	node1 *d_arr1;
	node2 *d_arr2;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	gpuErrchk(hipMalloc(&d_arr1,sizeof(node1)*N));
	gpuErrchk(hipMalloc(&d_arr2,sizeof(node2)));
	
	i=ceil(N/1024)+1;

	hipEventRecord(start);
	dKernel1<<<i,1024>>>(d_arr1);
	hipEventSynchronize(stop);
	hipEventRecord(stop);
	hipEventElapsedTime(&ms,start,stop);

	gpuErrchk( hipPeekAtLastError() );
	printf("TIME TAKEN BY AOS %f\n",ms);


	hipEventRecord(start);
	dKernel2<<<i,1024>>>(d_arr2);
	hipEventSynchronize(stop);
	hipEventRecord(stop);
	hipEventElapsedTime(&ms,start,stop);
	gpuErrchk( hipPeekAtLastError() );
	printf("TIME TAKEN BY SOA %f\n",ms);


	printf("EXITING\n");
	return 0;
}
