
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#define N 1000

__global__ void str_add(char *a, int len) {
	if(threadIdx.x < len)
		++a[threadIdx.x];
}

int main() {
	char *d_a;
//	char *str="Hello World";
	char str[]={'H', 'E', 'L', 'L', 'O', ' ', 'W', 'O', 'R', 'L', 'D'};

	hipMalloc(&d_a, strlen(str)*sizeof(char));

	hipMemcpy(d_a, str,  strlen(str)*sizeof(char), hipMemcpyHostToDevice);

	str_add<<<1,32>>>(d_a, strlen(str));

	hipMemcpy(str, d_a, strlen(str)*sizeof(char), hipMemcpyDeviceToHost);

	//COMPUTATIONS AFTER THE MEMCPY

	printf("%s\n",str);

	hipFree(d_a);

	return 0;
}
