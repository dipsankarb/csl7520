
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void printk(int *counter){

	do{	
		while(*counter % 2 == 0)
			printf("GPU COUNTER IS %d\n",*counter);
		++*counter;
	}while(*counter<1000000);
}

int main(){
	int hcounter=0,*counter;

//	cudaMalloc(&counter, sizeof(int));

	hipHostAlloc(&counter,sizeof(int),0);

	printk<<<1,1>>>(counter);

	do{
		while(*counter % 2 == 1)
			printf("CPU COUNTER IS %d \n",*counter);
//		cudaMemcpy(counter,&hcounter,sizeof(int),cudaMemcpyHostToDevice);
	//	cudaDeviceSynchronize();
//		cudaMemcpy(&hcounter,counter,sizeof(int),cudaMemcpyDeviceToHost);

	}while(++*counter<10);
	return 0;
}
