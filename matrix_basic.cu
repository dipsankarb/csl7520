#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#define N 512
#define M 10
#define in(i,j) ((i*N)+j)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void kernelV1(int *m){

	int tid;
	tid=blockIdx.x*blockDim.x+threadIdx.x;
	m[tid]=tid;
}

__global__ void squareGPUv1(int *m, int *res){
	int tid,j,k;
	tid=threadIdx.x+blockIdx.x*blockDim.x;
	for(j=0;j<blockDim.x;j++){
		for(k=0;k<blockDim.x;k++){
			res[in(tid,j)] += m[in(tid,k)]*m[in(k,j)];
		}
	}
	
}

__global__ void squareGPUv2(int *m, int *res){
	int tid,j,k,ii;

	tid=threadIdx.x+blockIdx.x*blockDim.x;
	j=tid%N;
	ii=tid/N;
	
	for(k=0;k<blockDim.x;k++){
			res[in(ii,j)] += m[in(ii,k)]*m[in(k,j)];
	}
}


void squareCPU(int *m, int *res){

	int i,j,k;

	for(i=0;i<N;i++){
		#pragma parallel for num_threads(16)
		for(j=0;j<N;j++){
			for(k=0;k<N;k++){
				res[in(i,j)] += m[in(i,k)]*m[in(k,j)];
			}
		}
	}

}
int main(){

	int *m, *d_m, *res, *d_res, i,j;
	clock_t start, stop;
	float ms;

	omp_set_num_threads(20);

	hipEvent_t s,p;

	hipEventCreate(&s);
	hipEventCreate(&p);

	m=(int*)malloc(N*N*sizeof(int));
	res=(int*)malloc(N*N*sizeof(int));
	hipMalloc(&d_m,sizeof(int)*N*N);
	hipMalloc(&d_res,sizeof(int)*N*N);

	kernelV1<<<N,N>>>(d_m);

	hipMemcpy(m,d_m,N*N*sizeof(int),hipMemcpyDeviceToHost);

/*	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			printf("%d ",m[in(i,j)]);
		}
		printf("\n");
	}*/


	start=clock();
	squareCPU(m,res);
	stop=clock();

	printf("Time taken on CPU is %lf\n",((double)(stop-start)/CLOCKS_PER_SEC)*1e3);

/*	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			printf("%d ",res[in(i,j)]);
		}
		printf("\n");
	}*/

	hipEventRecord(s);

	squareGPUv1<<<1,N>>>(d_m,d_res);

	hipEventRecord(p);
	hipEventSynchronize(p);
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&ms,s,p);
	printf("Time taken on GPU v1 kernel is %f\n",ms);

	hipEventRecord(s);

	squareGPUv2<<<N,N>>>(d_m,d_res);

	hipEventRecord(p);
	hipEventSynchronize(p);
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&ms,s,p);
	printf("Time taken on GPU v2 kernel is %f\n",ms);


	return 0;
}
