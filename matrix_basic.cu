#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 128
#define M 10
#define in(i,j) ((i*N)+j)

__global__ void kernelV1(int *m){

	int tid;
	tid=blockIdx.x*blockDim.x+threadIdx.x;
	m[tid]=tid;
}

__global__ void squareGPUv1(int *m, int *res){
	int tid,j,k;
	tid=threadIdx.x+blockIdx.x*blockDim.x;
	for(j=0;j<blockDim.x;j++){
		for(k=0;k<blockDim.x;k++){
			res[in(tid,j)] += m[in(tid,k)]*m[in(k,j)];
			printf("IN HERE\n");
		}
	}
	
}

void squareCPU(int *m, int *res){

	int i,j,k;

	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			for(k=0;k<N;k++){
				res[in(i,j)] += m[in(i,k)]*m[in(k,j)];
			}
		}
	}

}
int main(){

	int *m, *d_m, *res, i,j;
	clock_t start, stop;
	float ms;

	hipEvent_t s,p;

	hipEventCreate(&s);
	hipEventCreate(&p);

	m=(int*)malloc(N*N*sizeof(int));
	res=(int*)malloc(N*N*sizeof(int));
	hipMalloc(&d_m,sizeof(int)*N*N);

	kernelV1<<<N,N>>>(d_m);

	hipMemcpy(m,d_m,N*N*sizeof(int),hipMemcpyDeviceToHost);

/*	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			printf("%d ",m[in(i,j)]);
		}
		printf("\n");
	}*/


	start=clock();
	squareCPU(m,res);
	stop=clock();

	printf("Time taken on CPU is %lf\n",((double)(stop-start)/CLOCKS_PER_SEC)*1e3);

/*	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			printf("%d ",res[in(i,j)]);
		}
		printf("\n");
	}*/

	hipEventRecord(s);

	squareGPUv1<<<1,N>>>(m,res);

	hipEventRecord(p);
	hipEventSynchronize(p);

	hipEventElapsedTime(&ms,s,p);
	printf("Time taken on GPU v1 kernel is %f\n",ms);

	return 0;
}
