
#include <hip/hip_runtime.h>
#include<stdio.h>
#define BLOCK 1024

__global__ void dKernel(int *a){
	__shared__ int s;

	if(threadIdx.x == 0) s=0; // WARP X
	__syncthreads();

	if(threadIdx.x == 1) s+=1; // WARP Y
	__syncthreads();

	if(threadIdx.x == 100) s+=2; //WARP Z
	__syncthreads();

	//if(threadIdx.x == 0) *a=s; // WARP K
	if(threadIdx.x == 0) printf("S IS %d\n",s);; // WARP K
}
int main(){

	int *d_a,a,i;
	hipMalloc(&d_a,sizeof(int));

	for(i=0;i<100;i++){
		dKernel<<<2,BLOCK>>>(d_a);
		hipDeviceSynchronize();
	}

	hipMemcpy(&a,d_a,sizeof(int),hipMemcpyDeviceToHost);
//	printf("S IS %d\n",a);

	return 0;
}
