
#include <hip/hip_runtime.h>
#include<stdio.h>
#define BLOCK 1024

__global__ void deviceKernel(){
	__shared__ unsigned s;

	if(threadIdx.x==0) s=0;

	__syncthreads();

	if(threadIdx.x==1) s+=1;

	__syncthreads();

	if(threadIdx.x==100) s+=2;

	__syncthreads();

	if(threadIdx.x == 0) printf("s is %d\n",s);
}

int main(){
	int i;
	
	for(i=0;i<10;i++){
		deviceKernel<<<2,BLOCK>>>();
		hipDeviceSynchronize();
	}
	return 0;
}
