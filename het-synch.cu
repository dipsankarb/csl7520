
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void printk(int *counter){

	++*counter;
	printf("GPU COUNTER IS %d\n",*counter);
}

int main(){
	int hcounter=0,*counter;

//	cudaMalloc(&counter, sizeof(int));

	hipHostAlloc(&counter,sizeof(int),0);

	do{
		printf("CPU COUNTER IS %d \n",*counter);
//		cudaMemcpy(counter,&hcounter,sizeof(int),cudaMemcpyHostToDevice);
		printk<<<1,1>>>(counter);
		hipDeviceSynchronize();
//		cudaMemcpy(&hcounter,counter,sizeof(int),cudaMemcpyDeviceToHost);

	}while(++*counter<10);
	return 0;
}
