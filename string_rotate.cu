#include "hip/hip_runtime.h"

#include<stdio.h>
#define BLOCK 26

__global__ void deviceKernel(){

	int id;
	id=blockIdx.x*blockDim.x+threadIdx.x;

	char str[27];
	
	str[threadIdx.x]='A'+(blockIdx.x+threadIdx.x)%26;

	str[26]='\0';

	__syncthreads();

	if(threadIdx.x==0)
		printf("%s\n",str);
}	

int main(){
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(deviceKernel), hipFuncCachePreferEqual);

	deviceKernel<<<26, BLOCK>>>();

	hipDeviceSynchronize();


	return 0;
}
