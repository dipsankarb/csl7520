
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *x){
	*x=0;
	printf("%d\n",*x);
}

int main(){

	int *x;
	kernel<<<2,10>>>(x);
	hipDeviceSynchronize();

	//cudaError_t err = cudaGetLastError();
	//printf("error is %d, %s, %s\n",err,cudaGetErrorName(err),cudaGetErrorString(err));

	return 0;
}
