

#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 1024
#define BLOCK 1024

__global__ void reduceV1(int *elems){
	int id,i;
	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(i=N/2; i; i/=2) {
		if(id<i)
			elems[id] += elems[id+i];
		__syncthreads();
	}
	if(id==0)
		printf("GPU Sum is %d\n",elems[0]);
}

int main(){
	int host[N],i;
	long int sum=0;

	for(i=0;i<N;i++){
		host[i]=rand()%20;
		sum+=host[i];
	}	

	printf("CPU Sum is %d\n",sum);

	int *d_elems;
	hipMalloc(&d_elems,N*sizeof(int));

	hipMemcpy(d_elems,host,N*sizeof(int),hipMemcpyHostToDevice);

	reduceV1<<<(N+BLOCK-1)/BLOCK,BLOCK>>>(d_elems);
	hipDeviceSynchronize();

	return 0;
}
