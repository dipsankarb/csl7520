#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 1025

__global__ void init(int *a, int *b) {
	int id=threadIdx.x;
	a[id]=0;
	b[id]+=1;
}

__global__ void assign(int *a){
	int id=threadIdx.x;
	a[id]=id;
}

int main() {
	int a[N], i;

	int *d_a;

	hipMalloc(&d_a, N*sizeof(int));

	init<<<1,N>>>(d_a);

	hipDeviceSynchronize();

	assign<<<1,N>>>(d_a);


	hipMemcpy(a, d_a, sizeof(int)*N, hipMemcpyDeviceToHost);

	for(i=0; i<N; i++)
		printf("%d ",a[i]);

	return 0;
}
