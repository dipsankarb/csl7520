
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 1000

__global__ void init(int *a, int *b) {
	int id=threadIdx.x;
	a[id]=0;
	b[id]+=1;
}

__global__ void assign(int *a){
	int id=threadIdx.x;
	a[id]=id;
}

__global__ void assign_new(int *a){
	int id=threadIdx.x;
	int i;

	//NEW LOGIC
	#pragma unroll
	for(i=0; i<100; i++){
		a[id*100+i]=id*100+i;
	}

	
}



int main() {
	int a[N], i;

	int *d_a;

	hipMalloc(&d_a, N*sizeof(int));

	//init<<<1,N>>>(d_a);

	hipDeviceSynchronize();

	//assign<<<1,N>>>(d_a);
	
	assign_new<<<1,10>>>(d_a);


	hipMemcpy(a, d_a, sizeof(int)*N, hipMemcpyDeviceToHost);

	for(i=0; i<N; i++)
		printf("%d ",a[i]);

	return 0;
}
