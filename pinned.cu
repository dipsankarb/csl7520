
#include <hip/hip_runtime.h>
#include<stdio.h>

__device__ int global=0;

__host__ __device__ void add(){
	printf("This can run both on CPU and GPU\n");
	printf("GLOBAL FROM HOST DEVICE IS %d\n",global++);
}

__device__ void dkernel2(){
	printf("I am ANOTEHR device function\n");
	global++;
}


__device__ void dkernel(){
	printf("I am a device function\n");
	add();
	dkernel2();
	global++;
}

__global__ void kernel(int *c){
	++*c;
	printf("Counter on GPU %d\n",*c);
	dkernel();
	global++;
}

int main(){
	int *counter;

	hipHostMalloc(&counter, sizeof(int), 0, hipHostMallocDefault);
	
	*counter=0;

	add();

	do{
		printf("Counter on CPU %d\n",*counter);
		kernel<<<1,1>>>(counter);
		hipDeviceSynchronize();
		++*counter;
	}while(*counter<10);

	//printf("GLOBAL IS %d\n",global);

	return 0;
}
