#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime_api.h>

#define N 128

__global__ void squareMatrixGpu(double *a,double*b){
	int id,j,k;
	id = blockIdx.x*blockDim.x+threadIdx.x;
	for(j=0;j<N;j++){
		for(k=0;k<N;k++){
			b[id*N+j] += a[id*N+k]*a[id+N+j];
		}
	}
}

__global__ void squareMatrixGpuV2(double *a,double*b){
	int id,j,k,i;
	id = blockIdx.x*blockDim.x+threadIdx.x;
	i=id/N;
	j=id%N;
	for(k=0;k<N;k++){
		b[i*N+j] += a[i*N+k]*a[i+N+j];
	}
}


void squareMatrix(double *a, double *b){

	int i,j,k;

	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			for(k=0;k<N;k++){
				b[i*N+j]+=a[i*N+k]*a[k*N+j];
			}
		}
	}
}

int main() {

	double *a=(double*)malloc(N*N*sizeof(double));
	double *result=(double*)malloc(N*N*sizeof(double));

	double *d_a,*d_result;
	float ms=0;
	clock_t start,stop;
	int i,j,k;

	hipEvent_t st,end;

	hipEventCreate(&st);
	hipEventCreate(&end);

	
	for(i=0;i<N*N;i++){
		a[i]=(double)rand()/RAND_MAX*2.0-1.0;
		result[i]=0;
	}
	
	start=clock();
	squareMatrix(a,result);
	stop=clock();

	printf("Time taken to multiply on CPU %lf ms\n",((double)(stop-start)/CLOCKS_PER_SEC)*1000);

	hipMalloc(&d_a,N*N*sizeof(double));
	hipMalloc(&d_result,N*N*sizeof(double));

	hipProfilerStart();

	hipMemcpy(d_a,a,N*N*sizeof(double),hipMemcpyHostToDevice);

	hipEventRecord(st,0);

	squareMatrixGpu<<<1,N>>>(d_a,d_result);

	hipProfilerStop();
//	hipDeviceSynchronize();

	hipEventRecord(end);
	hipEventSynchronize(end);

	hipEventElapsedTime(&ms,st,end);

	printf("Time taken to multiply on GPU %f ms\n",ms);

	hipEventRecord(st,0);

	squareMatrixGpuV2<<<N,N>>>(d_a,d_result);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&ms,st,end);

	printf("Time taken to multiply on GPU-V2 %f ms\n",ms);

	return 0;
}
