

#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void function(){

	printf("Hello World! from Thread %d \n", threadIdx.x);
}

int main(){

	function<<<1,1025>>>(); // first two parameters in kernel calls are the thread configurations

	gpuErrchk( hipPeekAtLastError() );

	hipDeviceSynchronize();

	return 0;
}
