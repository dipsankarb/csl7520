
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

__device__ void deviceFunc(){

}

__global__ void deviceKernel(){
	printf("Hello World");
}

__global__ void gpuDoSomething(){
	int i;
	//for(i=0; i<N; i++)
	//	printf("%d\n",i*i);

	printf("%d\n",threadIdx.x*threadIdx.x);
}

void doSomething()
{
	int i;
	for(i=0; i<N; i++){
		printf("%d\n",i*i);
	}
}

int main(){
	//deviceKernel<<<1,32>>>();
	
	//cudaDeviceSynchronize();

	doSomething();

	printf("CALL GPU DO SOMETHING\n");

	gpuDoSomething<<<1, N>>>();

	hipDeviceSynchronize();

	return 0;
}
