
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initToZero(int *a){
	a[threadIdx.x]=threadIdx.x;
}

__global__ void addOne(int * d_a){
	d_a[threadIdx.x] += threadIdx.x;
}

__global__ void newAddOne(int *a){

	int tid;
	tid=blockIdx.x*blockDim.x+threadIdx.x;

	if((tid > 8000) || (tid < 1024))
		return;
	
	printf("Thread ID here is %d\n",tid);
	a[tid]+=tid;

}

int main(){

	int *a, *d_a;

	a=(int*)malloc(8000*sizeof(int));

	hipMalloc(&d_a,32*sizeof(int));

//	cudaMemcpy(d_a, a, sizeof(int)*32, cudaMemcpyHostToDevice);

	initToZero<<<1,32>>>(d_a);

//	cudaDeviceSynchronize();

	hipMemcpy(a, d_a, sizeof(int)*32, hipMemcpyDeviceToHost);

	hipFree(d_a);
	
	hipMalloc(&d_a,1024*sizeof(int));
	
	addOne<<<1,1024>>>(d_a);

	hipMemcpy(a, d_a, sizeof(int)*1024, hipMemcpyDeviceToHost);


	hipFree(d_a);

	hipMalloc(&d_a,8000*sizeof(int));

	hipMemcpy(a, d_a, sizeof(int)*1024, hipMemcpyDeviceToHost);

	newAddOne<<<ceil((8000-1024)/1024),1024>>>(d_a);

	hipMemcpy(a, d_a, sizeof(int)*8000, hipMemcpyDeviceToHost);

	printf("CHECK A[7999] %d\n",a[7999]);
	return 0;
}
