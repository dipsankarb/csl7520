
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void dkernel(unsigned int *x){
	++x[0]; // LEAD TO 1
	//atomicInc(&x[0],1000); // LEAD TO 2
}

int main() {
	int hx[]={0,0,0,0,0,0,0,0,0,0};
	unsigned int *x;
	hipMalloc(&x,sizeof(unsigned int)*10);
	hipMemcpy(x,hx,10*sizeof(unsigned int),hipMemcpyHostToDevice);
	
	dkernel<<<1,200>>>(x);
	
	hipDeviceSynchronize();
	hipMemcpy(hx,x,10*sizeof(unsigned int),hipMemcpyDeviceToHost);

	printf("x[0] is %d\n",hx[0]);

	return 0;
}
