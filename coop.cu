#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>

__device__ int k2, preprocess, work;

__global__ void kinit(){
	k2=0;
	preprocess=0;
}

__device__ void process(){
	int a[10]={0};
	for(int i=0;i<10;i++)
		a[i%2]++;
}

__global__ void firstkernel(int *work){
	process();
	__syncthreads;

	preprocess=1;
	*work=100;
	if(k2){
		*work /= 2;
	}
}

__global__ void secondkernel(int *work){
	if(preprocess)
		;
	else{
		k2=1;
		*work=100/2;
	}
}


int main(){
	srand(time(NULL));
	hipStream_t s1, s2;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);

	int *work;
	hipMalloc((int**)&work,sizeof(int));
	
	for(int i=0;i<10;i++){
		kinit<<<1,1>>>();
		hipDeviceSynchronize();

		firstkernel<<<1,64,0,s1>>>(work);
		hipDeviceSynchronize();

		if(rand() % 2){
			secondkernel<<<1,64,0,s2>>>(work);
			printf("Two kernels working \n");
		}
		else
			printf("One kernel working\n");

		hipDeviceSynchronize();

		int h;
		hipMemcpy(&h, (int*)work, sizeof(int),hipMemcpyDeviceToHost);
		printf("Work is %d\n",h);
	}



	return 0;
}
